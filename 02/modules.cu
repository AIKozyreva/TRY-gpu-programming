#include "hip/hip_runtime.h"
#include "util.h"
#include "modules.h"

__global__ void relu(const float* input, float* output, int height, int width, int channels) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int nx = blockDim.x * gridDim.x;
    int total_features = height * width * channels;

    for (int out_x = x; out_x < total_features; out_x += nx) {
        output[out_x] = max(input[out_x], 0.0f);
    }
}

void ReLU::forward(const float* input_device, float* output_device, int height, int width, int channels) const {
    int num_blocks = (height * width * channels + BLOCK_SZ_1D - 1) / BLOCK_SZ_1D;
    relu<<<num_blocks, BLOCK_SZ_1D>>>(input_device, output_device, height, width, channels);
}

__global__ void sigmoid(const float* input, float* output, int height, int width, int channels) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int nx = blockDim.x * gridDim.x;
    int total_features = height * width * channels;

    for (int out_x = x; out_x < total_features; out_x += nx) {
        output[out_x] = 1 / (1 + exp(-input[out_x]));
    }
}

void Sigmoid::forward(const float* input_device, float* output_device, int height, int width, int channels) const {
    int num_blocks = (height * width * channels + BLOCK_SZ_1D - 1) / BLOCK_SZ_1D;
    sigmoid<<<num_blocks, BLOCK_SZ_1D>>>(input_device, output_device, height, width, channels);
}

__global__ void maxpool(const float* input, float* output, int height, int width, int channels) {
    // coordinates of the first pixel to process
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;
    // number of processed pixels by step
    int nx = blockDim.x * gridDim.x;
    int ny = blockDim.y * gridDim.y;
    int nz = blockDim.z * gridDim.z;

    int h_2 = (height + 1) / 2;
    int w_2 = (width + 1) / 2;

    // loop replaces if statement, in most cases calculated once
    for (int out_z = z; out_z < channels; out_z += nz) {
        for (int out_y = y; out_y < h_2; out_y += ny) {
            for (int out_x = x; out_x < w_2; out_x += nx) {
                int c00 = out_z * height * width + (out_y * 2) * width + (out_x * 2);
                int c01 = c00 + 1;
                int c10 = c00 + width;
                int c11 = c10 + 1;
                if (out_x * 2 + 1 >= width) {
                    c01 = c00;
                    c11 = c10;
                }
                if (out_y * 2 + 1 >= height) {
                    c10 = c00;
                    c11 = c01;
                }
                int linear_idx = out_z * h_2 * w_2 + out_y * w_2 + out_x;
                output[linear_idx] = max(max(max(input[c00], input[c01]), input[c10]), input[c11]);
            }
        }
    }
}

void MaxPool2d::forward(const float* input_device, float* output_device, int height, int width, int channels) const {
    dim3 grid_dim((width + BLOCK_SZ_2D - 1) / BLOCK_SZ_2D, (height + BLOCK_SZ_2D - 1) / BLOCK_SZ_2D);
    dim3 block_dim(BLOCK_SZ_2D, BLOCK_SZ_2D);
    maxpool<<<grid_dim, block_dim>>>(input_device, output_device, height, width, channels);
}
