#include "hip/hip_runtime.h"
#define STB_IMAGE_IMPLEMENTATION
#define STBI_FAILURE_USERMSG
#include "stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

#include <iostream>
#include <fstream>
#include <cstdint>
#include <cstdlib>

#define RED         0
#define GREEN       1
#define BLUE        2
#define NUM_COLORS  3

#define Y_RED   0.2125f
#define Y_GREEN 0.7154f
#define Y_BLUE  0.0721f

#define Y_LEVELS 256
#define BLOCK_SZ 32

void rgb2gray_CPU(const uint8_t* rgb_image, uint8_t* gray_image, int height, int width) {
    for (int i = 0; i < height; ++i) {
        for (int j = 0; j < width; ++j) {
            *gray_image++ = Y_RED * rgb_image[RED] + Y_GREEN * rgb_image[GREEN] + Y_BLUE * rgb_image[BLUE];
            rgb_image += NUM_COLORS;
        }
    }
}

__global__ void rgb2gray_GPU(const uint8_t* rgb_image, uint8_t* gray_image, int height, int width) {
    // coordinates of the first pixel to process
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    // number of processed pixels by step
    int nx = blockDim.x * gridDim.x;
    int ny = blockDim.y * gridDim.y;

    for (int i = y; i < height; i += ny) {
        for (int j = x; j < width; j += nx) {
            int linear_idx = i * width + j;
            const uint8_t* in_pixel = rgb_image + NUM_COLORS * linear_idx;
            uint8_t* out_pixel = gray_image + linear_idx;

            *out_pixel = Y_RED * in_pixel[RED] + Y_GREEN * in_pixel[GREEN] + Y_BLUE * in_pixel[BLUE];
        }
    }
}

void histogram_CPU(const uint8_t* gray_img, int* hist, int height, int width) {
    for (int i = 0; i < height; ++i)
        for (int j = 0; j < width; ++j)
            ++hist[*gray_img++];
}

__global__ void histogram_local_GPU(const uint8_t *gray_img, int* all_hists, int height, int width) {
    int t = threadIdx.y * blockDim.x + threadIdx.x; // thread linear idx in block
    int num_threads = blockDim.x * blockDim.y;      // thread total count in block
    
    // initialize local histogram for one block
    __shared__ int local_hist[Y_LEVELS];
    for (int i = t; i < Y_LEVELS; i += num_threads)
        local_hist[i] = 0;
    __syncthreads();

    // coordinates of the first pixel to process
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    // number of processed pixels by step
    int nx = blockDim.x * gridDim.x;
    int ny = blockDim.y * gridDim.y;

    for (int i = y; i < height; i += ny)
        for (int j = x; j < width; j += nx)
            atomicAdd(local_hist + gray_img[i * width + j], 1);
    __syncthreads();

    // copy local hist to global memory
    all_hists += (blockIdx.y * gridDim.x + blockIdx.x) * Y_LEVELS;
    for (int i = t; i < Y_LEVELS; i += num_threads)
        all_hists[i] = local_hist[i];
}

__global__ void histogram_final_GPU(const int *all_hists, int *hist, int num_hists) {
    int t = blockIdx.x * blockDim.x + threadIdx.x; // thread global idx
    if (t < Y_LEVELS) {
        int total = 0;
        for (int i = 0; i < num_hists; i++) 
            total += all_hists[i * Y_LEVELS + t];
        hist[t] = total;
    }
}

void create_mapper(const int* hist, float* scaling_coeff, int pixel_count) {
    int cumsum[Y_LEVELS] = {};
    cumsum[0] = hist[0];
    scaling_coeff[0] = 0;
    for (int i = 1; i < Y_LEVELS; ++i) {
        cumsum[i] = cumsum[i-1] + hist[i];
        // mapper[i] = (Y_LEVELS * cumsum[i] + pixel_count - 1) / pixel_count - 1;
        scaling_coeff[i] = static_cast<float>((Y_LEVELS * cumsum[i] + pixel_count - 1) / pixel_count - 1) / i;
    }
}

void autocontrast_CPU(
    const uint8_t* rgb_src, uint8_t* rgb_dst,
    const uint8_t* gray_img, const float* scaling_coef,
    int height, int width, int channels)
{
    for (int i = 0; i < height; ++i) {
        for (int j = 0; j < width; ++j) {
            float coef = scaling_coef[*gray_img++];
            for (int k = 0; k < channels; ++k) { // RGB or Y
                *rgb_dst++ = std::min(*rgb_src++ * coef, 255.0f);
            }
        }
    }
}

__global__ void autocontrast_GPU(
    const uint8_t* rgb_src, uint8_t* rgb_dst,
    const uint8_t* gray_img, const float* scaling_coef,
    int height, int width, int channels)
{
    // coordinates of the first pixel to process
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    // number of processed pixels by step
    int nx = blockDim.x * gridDim.x;
    int ny = blockDim.y * gridDim.y;

    for (int i = y; i < height; i += ny) {
        for (int j = x; j < width; j += nx) {
            int linear_idx = i * width + j;
            const uint8_t* in_pixel = rgb_src + NUM_COLORS * linear_idx;
            uint8_t* out_pixel = rgb_dst + NUM_COLORS * linear_idx;
            float coef = scaling_coef[gray_img[linear_idx]];

            for (int k = 0; k < channels; ++k) { // RGB or Y
                *out_pixel++ = min(*in_pixel++ * coef, 255.0f);
            }
        }
    }
}

int main(int argc, char** argv) {
    if (argc < 3) {
        std::cout << "Usage: <input_image> <output_image>" << std::endl;
        return 0;
    }
    /// Load image
    int img_h, img_w, img_c;
    uint8_t* rgb_img = stbi_load(argv[1], &img_w, &img_h, &img_c, 0);
    if (!rgb_img) {
        std::cout << stbi_failure_reason() << std::endl;
        return 1;
    }
    std::cout << "Image loaded successfully. Shape: (" << img_h << ", " << img_w << ", " << img_c << ")" << std::endl;

    // Start processing. CPU
    // Allocate memory, initialize arrays
    uint8_t* gray_img = new uint8_t[img_h * img_w];
    uint8_t* res_img = new uint8_t[img_h * img_w * img_c];
    int histogram[Y_LEVELS] = {};
    float scaling_coeff[Y_LEVELS] = {};

    rgb2gray_CPU(rgb_img, gray_img, img_h, img_w);
#ifdef _DEBUG
    {
        int res = stbi_write_png("C:/Users/kosto/Desktop/work/gpu_programming/misc_files/_debug_grayscale_CPU.png", img_w, img_h, 1, gray_img, 0);
        if (!res) {
            std::cout << stbi_failure_reason() << std::endl;
            return 1;
        }
    }
#endif

    histogram_CPU(gray_img, histogram, img_h, img_w);
#ifdef _DEBUG
    {
        std::ofstream out_f("C:/Users/kosto/Desktop/work/gpu_programming/misc_files/_debug_hist_CPU.txt");
        for (int i = 0; i < Y_LEVELS; ++i) {
            out_f << histogram[i] << "\n";
        }
    }
#endif
    
    create_mapper(histogram, scaling_coeff, img_h * img_w);
#ifdef _DEBUG
    {
        std::ofstream out_f("C:/Users/kosto/Desktop/work/gpu_programming/misc_files/_debug_map_CPU.txt");
        for (int i = 0; i < Y_LEVELS; ++i) {
            out_f << scaling_coeff[i] << " " << scaling_coeff[i] * i << "\n";
        }
    }
#endif
    
#ifdef _DEBUG
    {
        autocontrast_CPU(gray_img, res_img, gray_img, scaling_coeff, img_h, img_w, 1);
        int res = stbi_write_png("C:/Users/kosto/Desktop/work/gpu_programming/misc_files/_debug_result_gray_CPU.png", img_w, img_h, 1, res_img, 0);
        if (!res) {
            std::cout << stbi_failure_reason() << std::endl;
            return 1;
        }
    }
#endif

    autocontrast_CPU(rgb_img, res_img, gray_img, scaling_coeff, img_h, img_w, img_c);
#ifdef _DEBUG
    {
        int res = stbi_write_png("C:/Users/kosto/Desktop/work/gpu_programming/misc_files/_debug_result_CPU.png", img_w, img_h, img_c, res_img, 0);
        if (!res) {
            std::cout << stbi_failure_reason() << std::endl;
            return 1;
        }
    }
#endif

    // Save image
    int res = stbi_write_png(argv[2], img_w, img_h, img_c, res_img, 0);
    if (!res) {
        std::cout << stbi_failure_reason() << std::endl;
        return 1;
    }

    // Start processing. GPU
    uint8_t *rgb_img_device, *gray_img_device, *res_img_device;
    int *all_hist_device, *histogram_device;
    float *scaling_coeff_device;
    dim3 grid_dim((img_w + BLOCK_SZ - 1) / BLOCK_SZ, (img_h + BLOCK_SZ - 1) / BLOCK_SZ);
    dim3 block_dim(BLOCK_SZ, BLOCK_SZ);

    hipMalloc(&rgb_img_device, img_h * img_w * img_c);
    hipMalloc(&gray_img_device, img_h * img_w);
    hipMalloc(&res_img_device, img_h * img_w * img_c);
    hipMalloc(&all_hist_device, Y_LEVELS * grid_dim.x * grid_dim.y * sizeof(int));
    hipMalloc(&histogram_device, Y_LEVELS * sizeof(int));
    hipMalloc(&scaling_coeff_device, Y_LEVELS * sizeof(float));

    hipMemcpy(rgb_img_device, rgb_img, img_h * img_w * img_c, hipMemcpyHostToDevice);
    rgb2gray_GPU<<<grid_dim, block_dim>>>(rgb_img_device, gray_img_device, img_h, img_w);
#ifdef _DEBUG
    {
        hipMemcpy(gray_img, gray_img_device, img_h * img_w, hipMemcpyDeviceToHost);
        int res = stbi_write_png("C:/Users/kosto/Desktop/work/gpu_programming/misc_files/_debug_grayscale_GPU.png", img_w, img_h, 1, gray_img, 0);
        if (!res) {
            std::cout << stbi_failure_reason() << std::endl;
            return 1;
        }
    }
#endif

    histogram_local_GPU<<<grid_dim, block_dim>>>(gray_img_device, all_hist_device, img_h, img_w);
    histogram_final_GPU<<<1, Y_LEVELS>>>(all_hist_device, histogram_device, grid_dim.x * grid_dim.y);
#ifdef _DEBUG
    {
        int debug_hist[Y_LEVELS] = {};
        hipMemcpy(debug_hist, histogram_device, Y_LEVELS * sizeof(int), hipMemcpyDeviceToHost);
        std::ofstream out_f("C:/Users/kosto/Desktop/work/gpu_programming/misc_files/_debug_hist_GPU.txt");
        for (int i = 0; i < Y_LEVELS; ++i) {
            out_f << debug_hist[i] << "\n";
        }
    }
#endif

    hipMemcpy(histogram, histogram_device, Y_LEVELS * sizeof(int), hipMemcpyDeviceToHost);
    create_mapper(histogram, scaling_coeff, img_h * img_w);
    hipMemcpy(scaling_coeff_device, scaling_coeff, Y_LEVELS * sizeof(float), hipMemcpyHostToDevice);

    autocontrast_GPU<<<grid_dim, block_dim>>>(rgb_img_device, res_img_device, gray_img_device, scaling_coeff_device, img_h, img_w, img_c);
    hipMemcpy(res_img, res_img_device, img_h * img_w * img_c, hipMemcpyDeviceToHost);
#ifdef _DEBUG
    {
        int res = stbi_write_png("C:/Users/kosto/Desktop/work/gpu_programming/misc_files/_debug_result_GPU.png", img_w, img_h, img_c, res_img, 0);
        if (!res) {
            std::cout << stbi_failure_reason() << std::endl;
            return 1;
        }
    }
#endif

    // Save image
    std::string out_fname(argv[2]);
    out_fname += "GPU.png";
    res = stbi_write_png(out_fname.c_str(), img_w, img_h, img_c, res_img, 0);
    if (!res) {
        std::cout << stbi_failure_reason() << std::endl;
        return 1;
    }



    return 0;
}
